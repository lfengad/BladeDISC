#include "hip/hip_runtime.h"

// arg0 = 25
// arg1 = [11776, 25]
// arg2 = 256
// arg3 = 376832
// arg4 = 4
// arg5 = [11776, 25]
// arg6 = 11776*25
// arg7 = 0
// arg8 = 0
// arg9 = 1
// arg10 = 1
// arg11 = 50
// arg12 = 11776
// arg13 = [11776, 50]
// arg14 = 0
// arg15 = 25
// arg16 = [11776, 25]
// arg17 = [11776, 25]
// arg18 = [11776, 25]
// arg19 = 11776
// arg20 = 25
// arg21 = [11776, 25]
// arg22 = [25]
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>

#define checkCudaErrors(val) \
	if (val != hipSuccess) exit(EXIT_FAILURE)


#ifndef D0
#define D0 11776
#endif
#ifndef D1
#define D1 25
#endif
#ifndef D2
#define D2 50
#endif
#ifndef LT
#define LT 256
#endif
#ifndef LB
#define LB (D0/(LT/LW)*LG/LI) // 376832
#endif
#ifndef LW
#define LW 8
#endif
#ifndef LG
#define LG 4
#endif
#ifndef LI
#define LI 1
#endif

template<typename T>
__global__ void kernel(T* arg1, T* arg5, T* arg16, T* arg17, T* arg18, T* arg13, T* arg21, T* arg22) {
     __shared__ T arg23[LT];
    int v14 = blockIdx.x * LT; //arg2 256 threaddim
    int v15 = threadIdx.x;
    int total_threads = v14 + v15;
    if (total_threads < LT*LB) {
        int thread_x = threadIdx.x; //v16 % 256;
        int v23 = threadIdx.x / LW; // v32
        int v24 = threadIdx.x % LW; // v32
        int v250 = blockIdx.x / LG;
        int v25 = v250 / LI;
        int v26 = blockIdx.x % LG;
        // int v28 = v25 * (LT/LW) + v23; // limit 368
        int v30 = v26 * (LW) + v24; // limit 32 -> 25
        bool v32 = v30 < D1;
        if (v32) {
          T local = 0.0;
          for (int i = 0; i < LI; i++) {
            int v28 = (v25 * LI + i) * (LT/LW) + v23;
            bool v31 = v28 < D0;
            if (v31 && v32) {
              int v53 = v28 * D1 + v30;
              T v55 = arg5[v53]; // total arg6
              T v64 = arg13[v28 * D2 + v30]; // total arg12 * arg11
              T v70 = arg13[v28 * D2 + v30 + D1];
              T v71 = v55 + v64 + v70;
              T v73 = arg1[v53]; //total arg6
              T v76 = arg16[v53];//total arg6
              T v78 = arg17[v53];//total arg6
              T v81 = -2 * v76 * v78 * v73 + v71;
              T v83 = arg18[v53]; // total arg6
              T v84 = v83 * v81;
              arg21[v53] = v84; // total arg19 * 
              local += v84;
            } 
          }
          arg23[thread_x] = local;
        } else {
          arg23[thread_x] = 0;
        }
        __syncthreads();
        int fac = LT;

        while(fac > LW * 2) {
          fac = fac/2;
          if (threadIdx.x < fac && ((v25 * LI) * (LT/LW) + v23) + fac / LW  < D0) {
                arg23[thread_x] = arg23[thread_x] + arg23[thread_x + fac];
            }

          __syncthreads();
        }
        //  fac = fac/2;
        // if (threadIdx.x < fac && v28 + fac / LW < D0) {
        //     arg23[thread_x] = arg23[thread_x] + arg23[thread_x + fac];
        // }
        // __syncthreads();
        //  fac = fac/2;
        // if (threadIdx.x < fac && v28 + fac / LW < D0) {
        //     arg23[thread_x] = arg23[thread_x] + arg23[thread_x + fac];
        // }
        // __syncthreads();
        //  fac = fac/2;
        // if (threadIdx.x < fac && v28 + fac / LW < D0) {
        //     arg23[thread_x] = arg23[thread_x] + arg23[thread_x + fac];
        // }
        // __syncthreads();
        if (v23 == 0 && (v25 * LI) * (LT/LW) + v23 < D0 && v32) {
            atomicAdd(&arg22[v30], arg23[thread_x] + arg23[thread_x + LW]);
        }
    }
}

// template __global__ void kernel<double>();


int main() {
  const int d0 = D0;
  const int d1 = D1;
  const int d2 = D2;
  double* ptr[6]; 
  double* output, *output0;
  uint64_t mem_size = d0 * d1 * sizeof(double);
  uint64_t mem_size0 = d0 * d2 * sizeof(double);
  uint64_t mem_size1 = d1 * sizeof(double);
  

  for (int i = 0; i < 5; i++) {
    checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&ptr[i]), mem_size));
  }
  checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&ptr[5]), mem_size0));

  checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&output), mem_size));
  checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&output0), mem_size1));
  double* h_init = reinterpret_cast<double*>(malloc(mem_size0));
  for (int64_t i = 0; i < mem_size0/sizeof(double) ; i++) {
      h_init[i] = 0.001;
   //   printf("%f ", h_init[i]);
  }
  for (int i = 0; i < 5; i++) {
    checkCudaErrors(hipMemcpy(ptr[i], h_init, mem_size, hipMemcpyDefault));
  }
  checkCudaErrors(hipMemcpy(ptr[5], h_init, mem_size0, hipMemcpyDefault));
//   printf("\n");
//   checkCudaErrors(hipMemcpy(input, h_init, mem_size, hipMemcpyDefault));
  checkCudaErrors(hipMemset(output, 0, mem_size));
  checkCudaErrors(hipMemset(output0, 0, mem_size1));
//   checkCudaErrors(hipMemset(mid, 0, mem_size));

//   free(h_init);
//   float yita = 0.0001;
//   float gamma = 1.3;
//   float beta = 1.3;

  hipEvent_t start, stop;
  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&stop));
  double* o_init = reinterpret_cast<double*>(malloc(mem_size1));
  memset(o_init, 0, mem_size1);

  // warmup
  checkCudaErrors(hipEventRecord(start));
  int times = 100;
  for (int i = 0; i < times; i++) {
	  /*
  	for (int64_t j = 0; j < mem_size/sizeof(float) ; j++) {
    		h_init[j] = 0.001*i;
  	}
        checkCudaErrors(hipMemcpy(input, h_init, mem_size, hipMemcpyDefault));*/
	kernel<double><<<LB, LT>>>(ptr[0], ptr[1], ptr[2], ptr[3], ptr[4], ptr[5], output, output0);
        // softmax1st_simple<<<d0, launch_dim>>>(input, output, d1);
	/*
  	checkCudaErrors(hipMemcpy(o_init, output, mem_size0, hipMemcpyDefault));*/
  }
  checkCudaErrors(hipEventRecord(stop));
  checkCudaErrors(hipEventSynchronize(stop));
  float msec = 0.0f;
  checkCudaErrors(hipEventElapsedTime(&msec, start, stop));
  printf("execution time = %f\n", msec / times);
//   float* o_mid = reinterpret_cast<float*>(malloc(mem_size));
//   memset(o_mid, 0, mem_size);
  checkCudaErrors(hipMemcpy(o_init, output0, mem_size1, hipMemcpyDefault));
   for (int64_t i = 0; i < mem_size1/sizeof(double); i++) {
 	 printf(" %f ", o_init[i]);
   }
   printf("\n");
//   for (int64_t i = 0; i < d0*d1; i++) {
// 	 printf(" %f ", o_mid[i]);
//   }
//   printf("\n");*/



  return 0;
}

















